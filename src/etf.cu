#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>

typedef int int_t;

__device__ int_t x;

// eulers totient funtion (commonly abbreviated as phi, here: etf)
// etf(1) = 1
// etf(n) = reduce (+) (map (coprime n) [1..n])
//
// the way to calculate it here is to use the amazing gpu cores for euclidean
// algorithm.

#define THREADS_PER_BLOCK 512
__device__ int_t coprime(int_t a, int_t b) {
	while(a > 0 && b > 0) {
		if(a == 1 || b == 1) {
			return 1;
		} else if(a > b) {
			a -= a / b * b;
		} else if(a < b) {
			b -= b / a * a;
		} else {
			if(a == 1)
				return 1;
			return 0;
		}
	}
	if(a == 1 || b == 1) {
		return 1;
	}
	return 0;
}

__global__ void kernel(int_t N) {
	__shared__ int_t mem[THREADS_PER_BLOCK];
	int tid = threadIdx.x;
	const int_t T = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x + 2;
	if(T > N)
		return;
	mem[tid] = coprime(N, T);
	__syncthreads();
	for(unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
		if(tid < s) {
			mem[tid] += mem[tid + s];
		}
		__syncthreads();
	}
	/* if(tid == 0) { */
	/* 	atomicAdd(&x, mem[0]); */
	/* } */
}

#ifdef NDEBUG
#define CUDACHK if(cuda_check() != EXIT_SUCCESS) return EXIT_FAILURE;
int cuda_check() {
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)  {
		fprintf(stderr, "error: %s\n", hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
#else
#define CUDACHK
#endif

int main(int argc, char *argv[]) {
	if(argc != 2 && argc != 3) {
		fputs("error: invalid number of arguments", stderr);
		return EXIT_FAILURE;
	}

	bool ranged = argc == 3;
	const int_t S = ranged ? atol(argv[1]) : 1;
	const int_t N = atol(argv[ranged ? 2 : 1]);
	if(S < 1 || N < S) {
		fputs("invalid domain", stderr);
		return EXIT_FAILURE;
	}

	int_t x_copy;
	static const int_t x_default = 1;
	if(S <= 1 && N >= 1)
		puts("1");
	for(int_t i = S + 1; i <= N; ++i) {
		hipMemcpy(x, &x_default, sizeof(int_t), hipMemcpyHostToDevice);
		CUDACHK;
		const int no_threads = i - 2;
		const int no_blocks = no_threads / THREADS_PER_BLOCK;
		kernel<<<no_blocks + 1, THREADS_PER_BLOCK>>>(i);
		CUDACHK;
		hipMemcpy(&x_copy, x, sizeof(int_t), hipMemcpyDeviceToHost);
		CUDACHK;
		printf("%d\n", x_copy);
	}
}
