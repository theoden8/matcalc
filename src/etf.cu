
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>

typedef int int_t;

__device__ int_t x;

#define THREADS_PER_BLOCK 512
__global__ void kernel(int_t N) {
	const int_t
		T = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x + 2;
	int_t a = N, b = T;
	/* printf("DEVICE [%d]: CALLED FOR (%d)\n", T, N); */
	if(T > N)
		return;
	/* printf("DEVICE [%d]: STARTED WITH VALUE (%d)\n", b, x); */
	while(a > 0 && b > 0) {
		if(a == 1 || b == 1) {
			/* printf("DEVICE: INCR\n"); */
			atomicAdd(&x, 1);
			return;
			/* goto end; */
		} else if(a > b) {
			a -= a / b * b;
		} else if(a < b) {
			b -= b / a * a;
		} else {
			if(a == 1) {
				/* printf("DEVICE: INCR\n"); */
				atomicAdd(&x, 1);
			}
			return;
			/* goto end; */
		}
	}
	if(a == 1 || b == 1) {
		/* printf("DEVICE: INCR\n"); */
		atomicAdd(&x, 1);
	}
/* end: */
/* 	; */
	/* printf("DEVICE: (%d, %d) FINISHED WITH RESULT %d\n", T, N, x); */
	/* __syncthreads(); */
	/* if(T + 1 == N) { */
	/* 	printf("%d\n", x); */
	/* } */
}

#ifdef NDEBUG
#define CUDACHK if(cuda_check() != EXIT_SUCCESS) return EXIT_FAILURE;
int cuda_check() {
	cudaError_t err = cudaGetLastError();
	if(err != cudaSuccess)  {
		fprintf(stderr, "error: %s\n", cudaGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
#else
#define CUDACHK
#endif

int main(int argc, char *argv[]) {
	if(argc != 2) {
		fputs("argc != 2", stderr);
		return EXIT_FAILURE;
	}

	const int_t N = atol(argv[1]);
	if(N < 1) {
		fputs("invalid domain", stderr);
		return EXIT_FAILURE;
	}

	int_t x_copy;
	static const int_t x_default = 1;
	if(N >= 1)
		puts("1");
	for(int_t i = 2; i <= N; ++i) {
		hipMemcpyToSymbol(HIP_SYMBOL(x), &x_default, sizeof(int_t));
		CUDACHK;
		const int no_threads = i - 2;
		const int no_blocks = no_threads / THREADS_PER_BLOCK;
		kernel<<<no_blocks + 1, THREADS_PER_BLOCK>>>(i);
		CUDACHK;
		hipMemcpyFromSymbol(&x_copy, HIP_SYMBOL(x), sizeof(int_t), 0, hipMemcpyDeviceToHost);
		CUDACHK;
		printf("%d\n", x_copy);
	}
}
